#include "hip/hip_runtime.h"
//
// Created by Anuj Chaudhari on 2018-12-08.
//

#include <cv.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include <string>
#include <unistd.h>
#include <ctime>
#include <libgen.h>
#include <math.h>

#include "../lib/dirreader.h"

using namespace cv;
using namespace std;

__device__
int TruncateDevice(int value) {
	if (value > 255)
		return 255;
	if (value < 0)
		return 0;
    return value;
}

int Truncate(int value) {
    if (value > 255)
        return 255;
    if (value < 0)
        return 0;
    return value;
}

uchar* convertImage(Mat mat) {
    // uchar *array = new uchar[mat.rows * mat.cols];
    uchar *array;
    if (mat.isContinuous())
        array = mat.data;
    return array;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__
void contrast_image(uchar* image, int rows, int cols, int factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    long index = (idx + idy * cols) * 3;

    if (idx >= cols || idy >= rows)
        return;

    image[index] = TruncateDevice( factor * ((int(image[index]) - 128) + 128) );
    image[index+1] = TruncateDevice( factor * ((int(image[index+1]) - 128) + 128) );
    image[index+2] = TruncateDevice( factor * ((int(image[index+2]) - 128) + 128) );
}

int main(int argc, char **argv) {
    int contrast = 127;

    vector<string> images;
    read_directory("./images/", images);

    double sequence_time = 0;
    double parallel_time = 0;

    for(int i = 0; i < images.size(); i++) {

        string inputFile = images[i];
        string input_filename = basename(strdup(inputFile.c_str()));

        cout << "Processing Image: " << inputFile << " : " << input_filename;

        Mat original_image = imread(inputFile, CV_LOAD_IMAGE_COLOR);
        uchar* image = convertImage(original_image);
        uchar* device_image;

        cout << "\t Image Resolution: " << original_image.rows << "x" << original_image.cols << endl;
        float factor = (259 * (contrast + 255)) / (255 * (259 - contrast));

        dim3 dimBlock(32,32);
        dim3 dimGrid;
        dimGrid.x = ceil(float(original_image.cols) / 32);
        dimGrid.y = ceil(float(original_image.rows) / 32);

        const clock_t begin_time = clock();

        gpuErrchk(hipMalloc((void**) &device_image, 3 *  original_image.rows * original_image.cols *sizeof(uchar))); 
        gpuErrchk(hipMemcpy(device_image, image, 3 * original_image.rows * original_image.cols *sizeof(uchar), hipMemcpyHostToDevice));
        contrast_image<<< dimGrid, dimBlock >>>(device_image, original_image.rows, original_image.cols, factor);
        gpuErrchk(hipMemcpy(image, device_image, 3 * original_image.rows * original_image.cols *sizeof(uchar), hipMemcpyDeviceToHost));
        gpuErrchk(hipFree(device_image));

        const clock_t end_time = clock();
        parallel_time += float( end_time - begin_time ) /  CLOCKS_PER_SEC;
        cout << "Par:" << float( end_time - begin_time ) /  CLOCKS_PER_SEC << endl;

        const clock_t begin_time_seq = clock();
        for (int i = 0; i < original_image.rows; i++) {
            for (int j = 0; j < original_image.cols * 3; j++) {
                image[j*original_image.rows+i] = Truncate(factor * (int(image[j*original_image.rows+i]-128)+128 ));
            }
        }
        const clock_t end_time_seq = clock();
        sequence_time += float( end_time_seq - begin_time_seq ) /  CLOCKS_PER_SEC;
        cout << "Seq: "<< float( end_time_seq - begin_time_seq ) /  CLOCKS_PER_SEC << endl;
        
        string output_filename = "./output/contrast_seq_" + input_filename;
        imwrite(output_filename, original_image);

        original_image.data = image;
        output_filename = "./output/contrast_cuda_" + input_filename;
        imwrite(output_filename, original_image);
    }

    cout << "Total Sequenctial Time: " << sequence_time << endl;
    cout << "Total Parallel    Time: " << parallel_time << endl;

    return 0;
}
