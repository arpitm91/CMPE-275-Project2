#include "hip/hip_runtime.h"
//
// Created by Anuj Chaudhari on 2018-12-08.
//

#include <cv.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include <string>
#include <unistd.h>
#include <ctime>

using namespace cv;
using namespace std;

int Truncate(int value) {
	if (value > 255)
		return 255;
	if (value < 0)
		return 0;
    return value;
}

uchar* convertImage(Mat mat) {
        uchar *array = new uchar[mat.rows * mat.cols];
            if (mat.isContinuous())
                            array = mat.data;
                return array;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__
void add(uchar* image, int rows, int cols, int factor)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < rows; i+=stride){
        for(int j = 0; j < 3*cols; j++) {
            int x = factor * ((int(image[j*rows+i])-128)+128);
            if (x > 255)
                x = 255;
           else if(x < 0)
                x = 0;
            image[j*rows+i] = x;
         }
    }
}

int main(int argc, char **argv) {
    int contrast = 127;
    int threads = 1;
    if (argc < 2) {
        printf("Usage: ./executable filename [contrast]\n");
        return -1;
    }
    if (argc >= 3) {
        threads = atoi(argv[2]);
    }

    string inputFile = argv[1];
    Mat original_image = imread(inputFile, CV_LOAD_IMAGE_COLOR);
    uchar* image = convertImage(original_image);
    uchar* device_image;
    uchar* returned_image = (uchar *) malloc(3 * original_image.rows * original_image.cols *sizeof(uchar));
//    uchar* returned_image;

    cout << "Image Resolution: " << original_image.rows << "x" << original_image.cols << endl;
    
    float factor = (259 * (contrast + 255)) / (255 * (259 - contrast));
    printf("Calling hipMallocManaged...\n");

    const clock_t begin_time = clock();

    hipMalloc((void**) &device_image, 3 *  original_image.rows * original_image.cols *sizeof(uchar));
    
    //printf("Calling cudaMemCpy H->D...\n");
    gpuErrchk( hipMemcpy(device_image, image, 3 * original_image.rows * original_image.cols *sizeof(uchar), hipMemcpyHostToDevice));

    //printf("Calling Kernel...\n");
    add<<<1, threads>>>(device_image, original_image.rows, original_image.cols, factor);
   
    //printf("Waiting for Device Synchronize...\n");
 //   hipDeviceSynchronize();
    //printf("Calling cudaMemCpy D->H...\n");

//    for(int i = 1000; i < 1010; i++)
//        cout<< int(returned_image[i]) << " " << int(image[i]) << endl;

    //cout << endl;
    gpuErrchk(hipMemcpy(returned_image, device_image, 3 * original_image.rows * original_image.cols *sizeof(uchar), hipMemcpyDeviceToHost));
    hipFree(device_image);

    cout << float( clock () - begin_time ) /  CLOCKS_PER_SEC;

    for (int i = 0; i < original_image.rows; i++) {
        for (int j = 0; j < original_image.cols * 3; j++) {
            image[j*original_image.rows+i] = Truncate(factor * (int(image[j*original_image.rows+i]-128)+128 ));
        }
    }
    
//    for(int i = 1000; i < 1010; i++)
//        cout<< int(returned_image[i]) << " " << int(image[i]) << endl;

    original_image.data = returned_image;

    imwrite("./output/contrast_cuda.jpg", original_image);
    return 0;
}
