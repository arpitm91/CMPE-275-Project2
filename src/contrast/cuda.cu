#include "hip/hip_runtime.h"
//
// Created by Anuj Chaudhari on 2018-12-08.
//

#include <cv.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include <string>
#include <unistd.h>
#include <ctime>
#include <math.h>
#include <sys/time.h>

using namespace cv;
using namespace std;

__device__
int TruncateDevice(int value) {
	if (value > 255)
		return 255;
	if (value < 0)
		return 0;
    return value;
}

int Truncate(int value) {
    if (value > 255)
        return 255;
    if (value < 0)
        return 0;
    return value;
}

uchar* convertImage(Mat mat) {
    uchar *array;
    if (mat.isContinuous())
        array = mat.data;
    return array;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__
void contrast_image(uchar* image, int rows, int cols, int factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    long index = (idx + idy * cols) * 3;

    if (idx >= cols || idy >= rows)
        return;

    image[index] = TruncateDevice( factor * ((int(image[index]) - 128) + 128) );
    image[index+1] = TruncateDevice( factor * ((int(image[index+1]) - 128) + 128) );
    image[index+2] = TruncateDevice( factor * ((int(image[index+2]) - 128) + 128) );
}

int main(int argc, char **argv) {
    int contrast = 127;
    if (argc < 2) {
        printf("Usage: ./executable filename [contrast]\n");
        return -1;
    }

    string inputFile = argv[1];
    Mat original_image = imread(inputFile, CV_LOAD_IMAGE_COLOR);
    uchar* image = convertImage(original_image);
    uchar* device_image;

    float factor = (259 * (contrast + 255)) / (255 * (259 - contrast));

    struct timeval start, end;
    gettimeofday(&start, NULL);
    
    dim3 dimBlock(32,32);
    dim3 dimGrid;
    dimGrid.x = ceil(float(original_image.cols) / 32);
    dimGrid.y = ceil(float(original_image.rows) / 32);

    gpuErrchk(hipMalloc((void**) &device_image, 3 *  original_image.rows * original_image.cols *sizeof(uchar))); 
    gpuErrchk(hipMemcpy(device_image, image, 3 * original_image.rows * original_image.cols *sizeof(uchar), hipMemcpyHostToDevice));
    contrast_image<<< dimGrid, dimBlock >>>(device_image, original_image.rows, original_image.cols, factor);
    gpuErrchk(hipMemcpy(image, device_image, 3 * original_image.rows * original_image.cols *sizeof(uchar), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(device_image));

    gettimeofday(&end, NULL);
    float delta = ((end.tv_sec  - start.tv_sec) * 1000000u +
                             end.tv_usec - start.tv_usec) / 1.e6;
    cout << delta;
    original_image.data = image;
    // imwrite("./output/contrast_cuda.jpg", original_image);
    return 0;
}
