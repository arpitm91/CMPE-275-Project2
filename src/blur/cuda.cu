#include "hip/hip_runtime.h"
//
// Created by Aartee Kasliwal on 2018-12-09.
//

#include<stdio.h>
#include <iostream>
#include <string>
#include <cv.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/gpu/gpu.hpp>

using namespace cv;
using namespace std;

const int GAUSSIAN_RADIUS = 11;

uchar* convertImage(Mat mat) {
	uchar *array = new uchar[mat.rows * mat.cols];
	if (mat.isContinuous())
		array = mat.data;
	return array;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__
int Truncate_device(int value) {
	if (value > 255)
		return 255;
	if (value < 0)
		return 0;
	return value;
}

__global__
void blur_image(uchar* device_image, uint* multiplication_matrix, int rows, int cols, int radius){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if(idx + 2 * radius > cols - 1 || idy + 2 * radius > rows - 1) {
		return;
	}

	double red = 0;
	double green = 0;
	double blue = 0;
	double sum = 0;

	for(int iy = 0; iy < 2 * radius + 1; iy++) {
		if(idy + iy + radius > rows - 1) {
			continue;
		}

		for (int ix = 0; ix < 2 * radius + 1; ix++) {
			if(idx + ix + radius > cols - 1) {
				continue;
			}

			int index = 3 * ((idx + ix + radius) + (idy + iy + radius) * cols);

			blue += double(device_image[index + 0]) * multiplication_matrix[ix + iy * radius];
			green += double(device_image[index + 1]) * multiplication_matrix[ix + iy * radius];
			red += double(device_image[index + 2]) * multiplication_matrix[ix + iy * radius];
			sum += multiplication_matrix[ix + iy * radius];
		}
	}

	int index = 3 * ((idx + 2 * radius) + (idy + 2 * radius) * cols);

	device_image[index + 0] = Truncate_device(blue / sum);
	device_image[index + 1] = Truncate_device(green / sum);
	device_image[index + 2] = Truncate_device(red / sum);

}

int main(int argc, const char** argv){

	if (argc < 2) {
		printf("Usage: ./executable originalImagePath\n");
		return -1;
	}

	//read the image
	string originalImagePath = argv[1];
	Mat originalImage = imread(originalImagePath, CV_LOAD_IMAGE_COLOR);
	Mat outputImage = originalImage.clone();

	//check whether the image is loaded or not
	if (!originalImage.data) {
		printf("Error : No Image Data.\n");
		return -1;
	}
	printf("Image resolution: %d * %d \n", originalImage.rows, originalImage.cols);


	uchar* host_image = convertImage(originalImage);
	uchar* device_image;
	uint* device_multiplication_matrix;


	gpuErrchk(hipMalloc((void**) &device_image, 3 * originalImage.rows * originalImage.cols * sizeof(uchar)));
	gpuErrchk(hipMemcpy(device_image, host_image, 3 * originalImage.rows * originalImage.cols * sizeof(uchar), hipMemcpyHostToDevice));

	int size = (2 * GAUSSIAN_RADIUS + 1) * (2 * GAUSSIAN_RADIUS + 1);
	int multiplication_matrix[size];

	for(int i = 0; i < size; i++){
		multiplication_matrix[i] = 1;
	}

	gpuErrchk(hipMalloc((void**) &device_multiplication_matrix, size * sizeof(uint)));
	gpuErrchk(hipMemcpy(device_multiplication_matrix, multiplication_matrix, size * sizeof(uint), hipMemcpyHostToDevice));

	dim3 dimGrid;
	dimGrid.x = ceil(float(originalImage.cols) / 32);
	dimGrid.y = ceil(float(originalImage.rows) / 32);
	dim3 dimBlock(32,32);

	blur_image<<< dimGrid, dimBlock >>>(device_image, device_multiplication_matrix, originalImage.rows, originalImage.cols, GAUSSIAN_RADIUS);

	gpuErrchk(hipMemcpy(host_image, device_image, 3 * originalImage.rows * originalImage.cols * sizeof(uchar), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(device_image));
	gpuErrchk(hipFree(device_multiplication_matrix));

	outputImage.data = host_image;
	imwrite("output/bluredimage_cuda.png", outputImage);
	return 0;
}
